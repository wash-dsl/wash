#include "hip/hip_runtime.h"
#include "wash.hpp"

#include "cstone/domain/domain.hpp"
#include "cstone/findneighbors.hpp"

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#if DIM != 3
#error "Only 3-dimensional vectors are supported at the moment"
#endif

namespace wash {
    // The internal simulation variables shouldn't be accessible by the user
    // By putting them inside an anonymous namespace, we ensure that they are only accessible in this source file
    namespace {
        uint64_t max_iterations;
        size_t particle_cnt;
        double box_xmin;
        double box_ymin;
        double box_zmin;
        double box_xmax;
        double box_ymax;
        double box_zmax;
        cstone::BoundaryType box_xtype;
        cstone::BoundaryType box_ytype;
        cstone::BoundaryType box_ztype;
        std::vector<std::unique_ptr<Kernel>> init_kernels;
        std::vector<std::unique_ptr<Kernel>> loop_kernels;
        NeighborsFuncT neighbors_kernel;
        std::function<unsigned(unsigned, unsigned)> neighbors_func;
        unsigned neighbors_max;
        std::vector<unsigned> neighbors_cnt;
        std::vector<unsigned> neighbors_data;
        std::unordered_map<std::string, double> variables;
        size_t force_cnt;
        std::unordered_map<std::string, size_t> force_map;
        std::array<std::vector<double>, MAX_FORCES> force_data;
        std::vector<Particle> particles;
        std::vector<Particle> local_particles;
        std::string simulation_name;
        std::string output_file_name;
        bool started;
    }

    int Particle::get_id() const { return (int)get_force_scalar("id"); }

    double Particle::get_density() const { return get_force_scalar("density"); }

    void Particle::set_density(const double density) { set_force_scalar("density", density); }

    double Particle::get_mass() const { return get_force_scalar("mass"); }

    void Particle::set_mass(const double mass) { set_force_scalar("mass", mass); }

    double Particle::get_smoothing_length() const { return get_force_scalar("smoothing_length"); }

    void Particle::set_smoothing_length(const double smoothing_length) {
        set_force_scalar("smoothing_length", smoothing_length);
    }

    SimulationVecT Particle::get_pos() const { return get_force_vector("pos"); }

    void Particle::set_pos(const SimulationVecT pos) { set_force_vector("pos", pos); }

    SimulationVecT Particle::get_vel() const { return get_force_vector("vel"); }

    void Particle::set_vel(const SimulationVecT vel) { set_force_vector("vel", vel); }

    SimulationVecT Particle::get_acc() const { return get_force_vector("acc"); }

    void Particle::set_acc(const SimulationVecT acc) { set_force_vector("acc", acc); }

    double Particle::get_force_scalar(const std::string& force) const {
        return force_data.at(force_map.at(force)).at(local_idx);
    }

    void Particle::set_force_scalar(const std::string& force, const double value) {
        force_data.at(force_map.at(force)).at(local_idx) = value;
    }

    SimulationVecT Particle::get_force_vector(const std::string& force) const {
        auto x = force_data.at(force_map.at(force + "_x")).at(local_idx);
        auto y = force_data.at(force_map.at(force + "_y")).at(local_idx);
        auto z = force_data.at(force_map.at(force + "_z")).at(local_idx);
        return SimulationVecT{x, y, z};
    }

    void Particle::set_force_vector(const std::string& force, const SimulationVecT value) {
        force_data.at(force_map.at(force + "_x")).at(local_idx) = value.at(0);
        force_data.at(force_map.at(force + "_y")).at(local_idx) = value.at(1);
        force_data.at(force_map.at(force + "_z")).at(local_idx) = value.at(2);
    }

    double Particle::get_vol() const { return get_mass() / get_density(); }

    std::vector<Particle> Particle::get_neighbors() const {
        // TODO: use iterators instead of allocating temporary vectors (some changes to the API may be required)
        // or reuse a temporary vector (since the max size is known), but keep it thread private
        unsigned count = neighbors_cnt.at(local_idx);
        std::vector<Particle> neighbors;
        neighbors.reserve(count);
        for (unsigned i = 0; i < count; i++) {
            neighbors.push_back(particles.at(neighbors_data.at(local_idx * neighbors_max + i)));
        }
        return neighbors;
    }

    unsigned Particle::recalculate_neighbors(unsigned max_count) const { return neighbors_func(local_idx, max_count); }

    bool Particle::operator==(const Particle other) const { return global_idx == other.global_idx; }

    bool Particle::operator!=(const Particle other) const { return !(*this == other); }

    void ForceKernel::exec() const {
#pragma omp parallel for
        for (auto& p : get_particles()) {
            // TODO: perhaps remove neighbors from ForceFuncT, since neighbors can be accessed directly from a particle
            func(p, p.get_neighbors());
        }
    }

    void UpdateKernel::exec() const {
#pragma omp parallel for
        for (auto& p : get_particles()) {
            func(p);
        }
    }

    void ReductionKernel::exec() const {
        double local_result;
        MPI_Op mpi_op;
        switch (reduce_op) {
        case ReduceOp::max:
            local_result = -std::numeric_limits<double>::infinity();
#pragma omp parallel for reduction(max : local_result)
            for (auto& p : get_particles()) {
                local_result = std::max(local_result, map_func(p));
            }
            mpi_op = MPI_MAX;
            break;

        case ReduceOp::min:
            local_result = std::numeric_limits<double>::infinity();
#pragma omp parallel for reduction(min : local_result)
            for (auto& p : get_particles()) {
                local_result = std::min(local_result, map_func(p));
            }
            mpi_op = MPI_MIN;
            break;

        case ReduceOp::sum:
            local_result = 0;
#pragma omp parallel for reduction(+ : local_result)
            for (auto& p : get_particles()) {
                local_result += map_func(p);
            }
            mpi_op = MPI_SUM;
            break;

        case ReduceOp::prod:
            local_result = 1;
#pragma omp parallel for reduction(* : local_result)
            for (auto& p : get_particles()) {
                local_result *= map_func(p);
            }
            mpi_op = MPI_PROD;
            break;

        default:
            assert(false);
        }

        double global_result;
        MPI_Allreduce(&local_result, &global_result, 1, MPI_DOUBLE, mpi_op, MPI_COMM_WORLD);
        set_variable(variable, global_result);
    }

    void VoidKernel::exec() const { func(); }

    uint64_t get_max_iterations() { return max_iterations; }

    void set_max_iterations(const uint64_t iterations) {
        assert(!started);
        max_iterations = iterations;
    }

    size_t get_particle_count() { return particle_cnt; }

    void set_particle_count(const size_t count) {
        assert(!started);
        particle_cnt = count;
    }

    void set_bounding_box(const double min, const double max, const bool periodic) {
        assert(!started);
        set_bounding_box(min, max, min, max, min, max, periodic, periodic, periodic);
    }

    void set_bounding_box(const double xmin, const double xmax, const double ymin, const double ymax, const double zmin,
                          const double zmax, const bool x_periodic, const bool y_periodic, const bool z_periodic) {
        assert(!started);
        box_xmin = xmin;
        box_xmax = xmax;
        box_ymin = ymin;
        box_ymax = ymax;
        box_zmin = zmin;
        box_zmax = zmax;
        box_xtype = x_periodic ? cstone::BoundaryType::periodic : cstone::BoundaryType::open;
        box_ytype = y_periodic ? cstone::BoundaryType::periodic : cstone::BoundaryType::open;
        box_ztype = z_periodic ? cstone::BoundaryType::periodic : cstone::BoundaryType::open;
    }

    void add_force_scalar(const std::string force) {
        assert(!started);
        assert(force_map.find(force) == force_map.end());
        assert(force_cnt < MAX_FORCES);
        force_map.emplace(force, force_cnt);
        force_cnt++;
    }

    void add_force_vector(const std::string force) {
        assert(!started);
        add_force_scalar(force + "_x");
        add_force_scalar(force + "_y");
        add_force_scalar(force + "_z");
    }

    void add_variable(const std::string variable, double init_value) {
        assert(!started);
        assert(variables.find(variable) == variables.end());
        variables.emplace(variable, init_value);
    }

    void add_init_update_kernel(const UpdateFuncT func) {
        assert(!started);
        init_kernels.push_back(std::make_unique<UpdateKernel>(func));
    }

    void add_init_void_kernel(const VoidFuncT func) {
        assert(!started);
        init_kernels.push_back(std::make_unique<VoidKernel>(func));
    }

    void add_force_kernel(const ForceFuncT func) {
        assert(!started);
        loop_kernels.push_back(std::make_unique<ForceKernel>(func));
    }

    void add_update_kernel(const UpdateFuncT func) {
        assert(!started);
        loop_kernels.push_back(std::make_unique<UpdateKernel>(func));
    }

    void add_reduction_kernel(const MapFuncT map_func, const ReduceOp reduce_op, const std::string variable) {
        assert(!started);
        loop_kernels.push_back(std::make_unique<ReductionKernel>(map_func, reduce_op, variable));
    }

    void add_void_kernel(const VoidFuncT func) {
        assert(!started);
        loop_kernels.push_back(std::make_unique<VoidKernel>(func));
    }

    void set_default_neighbor_search(const unsigned max_count) {
        assert(!started);
        neighbors_max = max_count;
        neighbors_kernel = [](Particle& p) { p.recalculate_neighbors(neighbors_max); };
    }

    void set_neighbor_search_kernel(const NeighborsFuncT func, const unsigned max_count) {
        assert(!started);
        neighbors_max = max_count;
        neighbors_kernel = func;
    }

    std::string get_simulation_name() { return simulation_name; }

    void set_simulation_name(const std::string name) {
        assert(!started);
        simulation_name = name;
    }

    std::string get_output_file_name() { return output_file_name; }

    void set_output_file_name(const std::string name) {
        assert(!started);
        output_file_name = name;
    }

    std::vector<std::string> get_forces_scalar() {
        std::vector<std::string> res;
        for (auto& p : force_map) {
            std::string force = p.first;
            size_t l = force.length();
            if (l < 2 || force[l - 2] != '_' || (force[l - 1] != 'x' && force[l - 1] != 'y' && force[l - 1] != 'z')) {
                res.push_back(force);
            }
        }
        return res;
    }

    std::vector<std::string> get_forces_vector() {
        std::vector<std::string> res;
        for (auto& p : force_map) {
            std::string force = p.first;
            size_t l = force.length();
            if (l >= 2 && force[l - 2] == '_' && force[l - 1] == 'x') {
                res.push_back(force.substr(0, l - 2));
            }
        }
        return res;
    }

    std::vector<std::string> get_variables() {
        std::vector<std::string> res;
        for (auto& p : variables) {
            res.push_back(p.first);
        }
        return res;
    }

    double get_variable(const std::string& variable) { return variables.at(variable); }

    void set_variable(const std::string& variable, const double value) { variables.at(variable) = value; }

    std::vector<Particle>& get_particles() {
        assert(started);
        return local_particles;
    }

    std::tuple<int, int> init_mpi() {
        int rank = 0;
        int n_ranks = 0;
        MPI_Init(NULL, NULL);
        MPI_Comm_rank(MPI_COMM_WORLD, &rank);
        MPI_Comm_size(MPI_COMM_WORLD, &n_ranks);
        return std::make_tuple(rank, n_ranks);
    }

    cstone::Domain<uint64_t, double, cstone::GpuTag> init_domain(int rank, int n_ranks, size_t num_particles) {
        uint64_t bucket_size_focus = 64;
        // we want about 100 global nodes per rank to decompose the domain with +-1% accuracy
        uint64_t bucket_size = std::max(bucket_size_focus, num_particles / (100 * n_ranks));
        float theta = 0.5f;
        return cstone::Domain<uint64_t, double, cstone::CpuTag>(
            rank, n_ranks, bucket_size, bucket_size_focus, theta,
            cstone::Box(box_xmin, box_xmax, box_ymin, box_ymax, box_zmin, box_zmax, box_xtype, box_ytype, box_ztype));
    }

    void recreate_particles(unsigned count_with_halos, size_t start_idx, size_t end_idx) {
        auto& id = force_data.at(force_map.at("id"));

        particles.clear();
        particles.reserve(count_with_halos);
        for (unsigned i = 0; i < count_with_halos; i++) {
            particles.emplace_back(id.at(i), i);
        }

        local_particles.clear();
        local_particles.reserve(end_idx - start_idx);
        for (unsigned i = start_idx; i < end_idx; i++) {
            local_particles.emplace_back(id.at(i), i);
        }
    }

    void sync_domain(cstone::Domain<uint64_t, double, cstone::CpuTag>& domain, std::vector<size_t>& keys,
                     std::vector<double>& s1, std::vector<double>& s2, std::vector<double>& s3) {
        auto& x = force_data.at(force_map.at("pos_x"));
        auto& y = force_data.at(force_map.at("pos_y"));
        auto& z = force_data.at(force_map.at("pos_z"));
        auto& h = force_data.at(force_map.at("smoothing_length"));

        domain.sync(keys, x, y, z, h, make_tuple<std::vector<double>, MAX_FORCES, MAX_FORCES - 4>(force_data),
                    std::tie(s1, s2, s3));
        domain.exchangeHalos(std::tie(force_data.at(force_map.at("id"))), s1, s2);

        recreate_particles(domain.nParticlesWithHalos(), domain.startIndex(), domain.endIndex());
        // TODO: don't have to allocate neighbors arrays for halo particles but it's easier for indexing
        neighbors_cnt.resize(domain.nParticlesWithHalos());
        neighbors_data.resize(domain.nParticlesWithHalos() * neighbors_max);
    }

    void start() {
        assert(particle_cnt > 0);
        assert(neighbors_max > 0);
        assert(box_xmax > box_xmin);
        assert(box_ymax > box_ymin);
        assert(box_zmax > box_zmin);

        // Add default forces
        // TODO: id should be a std::vector<size_t>
        add_force_scalar("id");
        add_force_scalar("density");
        add_force_scalar("mass");
        add_force_vector("vel");
        add_force_vector("acc");

        // Add position and smoothing length forces (must reside in the last 4 positions of force_data)
        force_cnt = MAX_FORCES - 4;
        add_force_scalar("smoothing_length");
        add_force_vector("pos");

        assert(!started);
        started = true;

        auto init0 = std::chrono::high_resolution_clock::now();

        // Initialize MPI
        auto [rank, n_ranks] = init_mpi();
        size_t first_id = particle_cnt * rank / n_ranks;
        size_t last_id = particle_cnt * (rank + 1) / n_ranks;
        unsigned local_count = last_id - first_id;

        // Resize data buffers
        for (auto& data : force_data) {
            data.resize(local_count);
        }
        auto& id = force_data.at(force_map.at("id"));
        for (unsigned i = 0; i < local_count; i++) {
            id.at(i) = first_id + i;
        }
        recreate_particles(local_count, 0, local_count);

        // Initialize IO
        auto& io = get_io();
        io.set_path(simulation_name, output_file_name);

        // Time for IO initialization
        auto init1 = std::chrono::high_resolution_clock::now();
        io.write_timings("data_io_setup", -1, diff_ms(init0, init1));

        size_t k_idx = 0;
        for (auto& k : init_kernels) {
            auto init_k0 = std::chrono::high_resolution_clock::now();

            k->exec();

            // Time for this initialisation kernel
            auto init_k1 = std::chrono::high_resolution_clock::now();
            io.write_timings("init_kernel_run", k_idx++, diff_ms(init_k0, init_k1));
        }

        // Time for initialisation kernels
        auto init2 = std::chrono::high_resolution_clock::now();
        io.write_timings("init_kernels", -1, diff_ms(init1, init2));

        // Initialize and sync domain
        std::vector<size_t> keys(local_count);

        thrust::device_vector<double> d_x       = x;
        thrust::device_vector<double> d_y       = y;
        thrust::device_vector<double> d_z       = z;
        thrust::device_vector<double> d_h       = h;
        thrust::device_vector<size_t> d_keys = keys;

        // std::vector<double> s1;
        // std::vector<double> s2;
        // std::vector<double> s3;

        thrust::device_vector<Real> s1, s2, s3;

        auto domain = init_domain(rank, n_ranks, particle_cnt);
        // TODO: detect which forces are changed in any init kernel and only sync those forces (remember to resize force
        // vectors that were not synced)
        sync_domain(domain, keys, s1, s2, s3);

        // Handle IO before first iteration
        io.handle_iteration(-1);

        // Time for IO iteration
        auto init3 = std::chrono::high_resolution_clock::now();
        io.write_timings("init_io", -1, diff_ms(init2, init3));

        for (uint64_t iter = 0; iter < max_iterations; iter++) {
            k_idx = 0;
            auto iter0 = std::chrono::high_resolution_clock::now();

            // TODO: don't sync temp forces that don't need to be preserved across iterations (but remember to resize
            // the vectors)
            sync_domain(domain, keys, s1, s2, s3);

            auto x_ptr = force_data.at(force_map.at("pos_x")).data();
            auto y_ptr = force_data.at(force_map.at("pos_y")).data();
            auto z_ptr = force_data.at(force_map.at("pos_z")).data();
            auto h_ptr = force_data.at(force_map.at("smoothing_length")).data();
            auto tree_view = domain.octreeProperties().nsView();
            auto box = domain.box();

            // TODO: temporary workaround so that x, y, z, h don't have to be global (won't be needed in the DSL
            // version)
            neighbors_func = [x_ptr, y_ptr, z_ptr, h_ptr, tree_view, box](unsigned i, unsigned max_count) {
                unsigned count = cstone::findNeighbors(i, x_ptr, y_ptr, z_ptr, h_ptr, tree_view, box, max_count,
                                                       neighbors_data.data() + i * neighbors_max);
                neighbors_cnt.at(i) = std::min(count, neighbors_max);
                return count;
            };

            // TODO: find neighbors after domain sync only when necessary
            for (auto& p : get_particles()) {
                neighbors_kernel(p);
            }

            for (auto& k : loop_kernels) {
                auto iter_k0 = std::chrono::high_resolution_clock::now();

                // TODO: detect dependencies between forces used in each kernel and only exchange what's needed
                domain.exchangeHalos(make_tuple<std::vector<double>, MAX_FORCES>(force_data), s1, s2);

                k->exec();

                // Time for this loop kernel
                auto iter_k1 = std::chrono::high_resolution_clock::now();
                io.write_timings("kernel_run", k_idx++, diff_ms(iter_k0, iter_k1));
            }

            // Time for full iteration
            auto iter1 = std::chrono::high_resolution_clock::now();
            io.write_timings("iteration_run", iter, diff_ms(iter0, iter1));

            // Handle IO after this iteration
            io.handle_iteration(iter);

            std::cout << "Finished iter " << iter << std::endl;

            // Time for IO iteration
            auto iter2 = std::chrono::high_resolution_clock::now();
            io.write_timings("iteration_io", iter, diff_ms(iter1, iter2));
        }

        MPI_Finalize();
    }

    double eucdist(const Particle& p, const Particle& q) {
        SimulationVecT diff = p.get_pos() - q.get_pos();
        return diff.magnitude();
    }
}
